#include "head.h"

//variable for cublas
hipblasStatus_t stat;
hipblasHandle_t handle=0;

float * A;
float * d_A;
float * b;
float * d_b;
float * C;
float * d_C;
float * x;
float * d_x;
float alpha, beta;

#define batchsize 1
int * PivotA;
int * infoA;
int * h_PivotA;
int * h_infoA;

float ** h_Apoint;
float ** d_Apoint;
float ** h_Cpoint;
float ** d_Cpoint;


void Allocate_Memory(){
	hipError_t Error;

	A = (float *)malloc(m*n*sizeof(float));
	b = (float *)malloc(n*sizeof(float));
	x = (float *)malloc(n*sizeof(float));
	C = (float *)malloc(m*n*sizeof(float));

	Error = hipMalloc((void **)&d_A, m*n*sizeof(float));
	printf("CUDA error(malloc d_A) = %s\n",hipGetErrorString(Error));
	Error = hipMalloc((void **)&d_b, n*sizeof(float));
        printf("CUDA error(malloc d_b) = %s\n",hipGetErrorString(Error));
	Error = hipMalloc((void **)&d_x, n*sizeof(float));
        printf("CUDA error(malloc d_x) = %s\n",hipGetErrorString(Error));
	Error = hipMalloc((void **)&d_C, m*n*sizeof(float));
        printf("CUDA error(malloc d_C) = %s\n",hipGetErrorString(Error));

	h_PivotA = (int *)malloc(n*batchsize*sizeof(int));
	h_infoA = (int *)malloc(batchsize*sizeof(int));
	Error = hipMalloc((void **)&PivotA, n*batchsize*sizeof(int));
        printf("CUDA error(malloc PivotA) = %s\n",hipGetErrorString(Error));
	Error = hipMalloc((void **)&infoA, batchsize*sizeof(int));
        printf("CUDA error(malloc infoA) = %s\n",hipGetErrorString(Error));

	h_Apoint = (float **)malloc(batchsize*sizeof(float*));
	Error = hipMalloc((void **)&d_Apoint, batchsize*sizeof(float*));
        printf("CUDA error(malloc d_Apoint) = %s\n",hipGetErrorString(Error));
	h_Cpoint = (float **)malloc(batchsize*sizeof(float*));
        Error = hipMalloc((void **)&d_Cpoint, batchsize*sizeof(float*));
        printf("CUDA error(malloc d_Cpoint) = %s\n",hipGetErrorString(Error));
}

void Init(){
	int i, j;
	for(j=0;j<n;j++){
		for(i=0;i<m;i++){
			A[i+j*m] = int(rand()%10);
		}
	}
	for(i=0;i<n;i++){
		b[i] = int(rand()%10);
		x[i] = 0.0;
	}
	for(i=0;i<m*n;i++){
		C[i] = 0.0;
	}
	alpha = 1.0, beta = 0.0;

	for(i=0;i<batchsize;i++){
		h_Apoint[0] = d_A + i*m*n;
		h_Cpoint[0] = (float *)((char*)d_C+i*((size_t)m*n)*sizeof(float));
	}
}

void Send_To_Device(){
	hipError_t Error;
	Error = hipMemcpy(d_A, A, m*n*sizeof(float), hipMemcpyHostToDevice);
	printf("CUDA error(memcpy A) = %s\n",hipGetErrorString(Error));
	Error = hipMemcpy(d_b, b, n*sizeof(float), hipMemcpyHostToDevice);
        printf("CUDA error(memcpy b) = %s\n",hipGetErrorString(Error));
	Error = hipMemcpy(d_C, C, m*n*sizeof(float), hipMemcpyHostToDevice);
        printf("CUDA error(memcpy C) = %s\n",hipGetErrorString(Error));

	Error = hipMemcpy(d_Apoint, h_Apoint, batchsize*sizeof(float*), hipMemcpyHostToDevice);
        printf("CUDA error(memcpy Apoint) = %s\n",hipGetErrorString(Error));
	Error = hipMemcpy(d_Cpoint, h_Cpoint, batchsize*sizeof(float*), hipMemcpyHostToDevice);
        printf("CUDA error(memcpy Cpoint) = %s\n",hipGetErrorString(Error));
}

void Call_GPUFunction(){
	stat = hipblasCreate(&handle);
	if (stat != HIPBLAS_STATUS_SUCCESS) {
		printf ("CUBLAS initialization failed\n");
	}

	hipblasSgetrfBatched( handle,
		m,
		d_Apoint,
		m,
		PivotA,
		infoA,
		batchsize);

	hipblasSgetriBatched( handle,
		m,
		(const float **)d_Apoint,
		m,
		PivotA,
		d_Cpoint,
		m,
		infoA,
		batchsize);

	hipblasSgemv(  handle,
                HIPBLAS_OP_N,
                m,
                n,
                &alpha,
                d_C,
                m,
                d_b,
                1,
                &beta,
                d_x,
                1);

	hipDeviceSynchronize();
}


void Send_To_Host(){
	hipError_t Error;
	Error = hipMemcpy(x, d_x, n*sizeof(float), hipMemcpyDeviceToHost);
        printf("CUDA error(memcpy d_x->x) = %s\n",hipGetErrorString(Error));

	Error = hipMemcpy(C, d_C, m*n*sizeof(float), hipMemcpyDeviceToHost);
        printf("CUDA error(memcpy d_C->C) = %s\n",hipGetErrorString(Error));
	Error = hipMemcpy(A, d_A, m*n*sizeof(float), hipMemcpyDeviceToHost);
        printf("CUDA error(memcpy d_A->A) = %s\n",hipGetErrorString(Error));
	Error = hipMemcpy(h_PivotA, PivotA, n*batchsize*sizeof(int), hipMemcpyDeviceToHost);
        printf("CUDA error(memcpy PivotA) = %s\n",hipGetErrorString(Error));
	Error = hipMemcpy(h_infoA, infoA, batchsize*sizeof(int), hipMemcpyDeviceToHost);
        printf("CUDA error(memcpy infoA) = %s\n",hipGetErrorString(Error));
}

void Free_Memory(){
	if (A) free(A);
        if (d_A) hipFree(d_A);
	if (b) free(b);
        if (d_b) hipFree(d_b);
	if (x) free(x);
        if (d_x) hipFree(d_x);
	if (C) free(C);
        if (d_C) hipFree(d_C);

	if (h_PivotA) free(h_PivotA);
	if (PivotA) hipFree(PivotA);
	if (h_infoA) free(h_infoA);
	if (infoA) hipFree(infoA);

	if (h_Apoint) free(h_Apoint);
	if (d_Apoint) hipFree(d_Apoint);
	if (h_Cpoint) free(h_Cpoint);
        if (d_Cpoint) hipFree(d_Cpoint);

        if (handle) hipblasDestroy(handle);

}

void Save_Result() {

        FILE *pFile;
        int i;

        // Save the matrix
	pFile = fopen("x.txt","w");
        // Save the vector x
        for (i = 0; i < n; i++) {
		fprintf(pFile, "%g\t", x[i]);
		fprintf(pFile, "\n");
        }
        fclose(pFile);
}

