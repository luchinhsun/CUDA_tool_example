#include "head.h"

//variable for cublas
hipblasStatus_t stat;
hipblasHandle_t handle=0;

float * A;
float * d_A;
float * B;
float * d_B;
float * C;
float * d_C;
float alpha, beta;

void Allocate_Memory(){
	hipError_t Error;

	A = (float *)malloc(m*k*sizeof(float));
	B = (float *)malloc(k*n*sizeof(float));
	C = (float *)malloc(m*n*sizeof(float));

	Error = hipMalloc((void **)&d_A, m*k*sizeof(float));
	printf("CUDA error(malloc d_A) = %s\n",hipGetErrorString(Error));
	Error = hipMalloc((void **)&d_B, k*n*sizeof(float));
        printf("CUDA error(malloc d_B) = %s\n",hipGetErrorString(Error));
	Error = hipMalloc((void **)&d_C, m*n*sizeof(float));
        printf("CUDA error(malloc d_C) = %s\n",hipGetErrorString(Error));
}

void Init(){
	int i, j;
	for(j=0;j<k;j++){
		for(i=0;i<m;i++){
			A[i+j*m] = int(rand()%10);
		}
	}
	for(j=0;j<n;j++){
		for(i=0;i<k;i++){
			B[i+j*k] = int(rand()%10);
		}
	}
	for(j=0;j<n;j++){
		for(i=0;i<m;i++){
			C[i+j*m] = 0.0;
		}
	}
	alpha = 1.0, beta = 0.0;
}

void Send_To_Device(){
	hipError_t Error;
	Error = hipMemcpy(d_A, A, m*k*sizeof(float), hipMemcpyHostToDevice);
	printf("CUDA error(memcpy A) = %s\n",hipGetErrorString(Error));
	Error = hipMemcpy(d_B, B, k*n*sizeof(float), hipMemcpyHostToDevice);
        printf("CUDA error(memcpy B) = %s\n",hipGetErrorString(Error));
	Error = hipMemcpy(d_C, C, m*n*sizeof(float), hipMemcpyHostToDevice);
        printf("CUDA error(memcpy C) = %s\n",hipGetErrorString(Error));
}

void Call_GPUFunction(){
	stat = hipblasCreate(&handle);
	if (stat != HIPBLAS_STATUS_SUCCESS) {
		printf ("CUBLAS initialization failed\n");
	}

	hipblasSgemm(  handle,
                HIPBLAS_OP_N,
                HIPBLAS_OP_N,
                m,
                n,
                k,
                &alpha,
                d_A,
                m,
                d_B,
                k,
                &beta,
                d_C,
                m);
	//hipDeviceSynchronize();
}


void Send_To_Host(){
	hipError_t Error;
	Error = hipMemcpy(C, d_C, m*n*sizeof(float), hipMemcpyDeviceToHost);
        printf("CUDA error(memcpy d_C->C) = %s\n",hipGetErrorString(Error));
}

void Free_Memory(){
	if (A) free(A);
        if (d_A) hipFree(d_A);
	if (B) free(B);
        if (d_B) hipFree(d_B);
	if (C) free(C);
        if (d_C) hipFree(d_C);

        if (handle) hipblasDestroy(handle);

}

void Save_Result() {

        FILE *pFile;
        int i, j;

        // Save the matrix
	pFile = fopen("C.txt","w");
        // Save the matrix C
        for (i = 0; i < m; i++) {
		for (j = 0; j < n; j++) {
                	fprintf(pFile, "%g\t", C[i*n+j]);
		}
		fprintf(pFile, "\n");
        }
        fclose(pFile);
}

