#include "head.h"

//variable for cublas
hipblasStatus_t stat;
hipblasHandle_t handle=0;

float * a;
float * d_a;
float * b;
float * d_b;
float * c;
float * d_c;

void Allocate_Memory(){
	hipError_t Error;

	a = (float *)malloc(n*sizeof(float));
	b = (float *)malloc(n*sizeof(float));
	c = (float *)malloc(1*sizeof(float));

	Error = hipMalloc((void **)&d_a, n*sizeof(float));
	printf("CUDA error(malloc d_a) = %s\n",hipGetErrorString(Error));
	Error = hipMalloc((void **)&d_b, n*sizeof(float));
        printf("CUDA error(malloc d_b) = %s\n",hipGetErrorString(Error));
	Error = hipMalloc((void **)&d_c, 1*sizeof(float));
        printf("CUDA error(malloc d_c) = %s\n",hipGetErrorString(Error));
}

void Init(){
	int i;
	for(i=0;i<n;i++){
		a[i] = i;//int(rand()%10);
	}
	for(i=0;i<n;i++){
		b[i] = int(rand()%10);
	}
	c[0] = 0.0;
}

void Send_To_Device(){
	hipError_t Error;
	Error = hipMemcpy(d_a, a, n*sizeof(float), hipMemcpyHostToDevice);
	printf("CUDA error(memcpy A) = %s\n",hipGetErrorString(Error));
	Error = hipMemcpy(d_b, b, n*sizeof(float), hipMemcpyHostToDevice);
        printf("CUDA error(memcpy b) = %s\n",hipGetErrorString(Error));
	Error = hipMemcpy(d_c, c, 1*sizeof(float), hipMemcpyHostToDevice);
        printf("CUDA error(memcpy c) = %s\n",hipGetErrorString(Error));
}

void Call_GPUFunction(){
	stat = hipblasCreate(&handle);
	if (stat != HIPBLAS_STATUS_SUCCESS) {
		printf ("CUBLAS initialization failed\n");
	}

	hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_DEVICE);

	hipblasSdot(  handle,
                n,
                d_a,
                1,
                d_b,
                1,
                d_c);
	//hipDeviceSynchronize();

}


void Send_To_Host(){
	hipError_t Error;
	Error = hipMemcpy(c, d_c, 1*sizeof(float), hipMemcpyDeviceToHost);
        printf("CUDA error(memcpy d_c->c) = %s\n",hipGetErrorString(Error));
}

void Free_Memory(){
	if (a) free(a);
        if (d_a) hipFree(d_a);
	if (b) free(b);
        if (d_b) hipFree(d_b);
	if (c) free(c);
        if (d_c) hipFree(d_c);

        if (handle) hipblasDestroy(handle);

}

void Save_Result() {

        FILE *pFile;
        int i;

        // Save the vector
	pFile = fopen("c.txt","w");
        // Save the vector c
        for (i = 0; i < 1; i++) {
		fprintf(pFile, "%g\t", c[i]);
        }
        fclose(pFile);
}

