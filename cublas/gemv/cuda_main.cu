#include "head.h"

//variable for cublas
hipblasStatus_t stat;
hipblasHandle_t handle=0;

float * A;
float * d_A;
float * b;
float * d_b;
float * c;
float * d_c;
float alpha, beta;

void Allocate_Memory(){
	hipError_t Error;

	A = (float *)malloc(m*n*sizeof(float));
	b = (float *)malloc(n*sizeof(float));
	c = (float *)malloc(m*sizeof(float));

	Error = hipMalloc((void **)&d_A, m*n*sizeof(float));
	printf("CUDA error(malloc d_A) = %s\n",hipGetErrorString(Error));
	Error = hipMalloc((void **)&d_b, n*sizeof(float));
        printf("CUDA error(malloc d_b) = %s\n",hipGetErrorString(Error));
	Error = hipMalloc((void **)&d_c, m*sizeof(float));
        printf("CUDA error(malloc d_c) = %s\n",hipGetErrorString(Error));
}

void Init(){
	int i, j;
	for(j=0;j<n;j++){
		for(i=0;i<m;i++){
			A[i+j*m] = i+j*m;//int(rand()%10);
		}
	}
	for(i=0;i<n;i++){
		b[i] = int(rand()%10);
	}
	for(i=0;i<m;i++){
		c[i] = 0.0;
	}
	alpha = 1.0, beta = 0.0;
}

void Send_To_Device(){
	hipError_t Error;
	Error = hipMemcpy(d_A, A, m*n*sizeof(float), hipMemcpyHostToDevice);
	printf("CUDA error(memcpy A) = %s\n",hipGetErrorString(Error));
	Error = hipMemcpy(d_b, b, n*sizeof(float), hipMemcpyHostToDevice);
        printf("CUDA error(memcpy b) = %s\n",hipGetErrorString(Error));
	Error = hipMemcpy(d_c, c, m*sizeof(float), hipMemcpyHostToDevice);
        printf("CUDA error(memcpy c) = %s\n",hipGetErrorString(Error));
}

void Call_GPUFunction(){
	stat = hipblasCreate(&handle);
	if (stat != HIPBLAS_STATUS_SUCCESS) {
		printf ("CUBLAS initialization failed\n");
	}

	hipblasSgemv(  handle,
		HIPBLAS_OP_N,
                m,
                n,
                &alpha,
                d_A,
                m,
                d_b,
                1,
                &beta,
                d_c,
                1);
	//hipDeviceSynchronize();
}


void Send_To_Host(){
	hipError_t Error;
	Error = hipMemcpy(c, d_c, m*sizeof(float), hipMemcpyDeviceToHost);
        printf("CUDA error(memcpy d_c->c) = %s\n",hipGetErrorString(Error));
}

void Free_Memory(){
	if (A) free(A);
        if (d_A) hipFree(d_A);
	if (b) free(b);
        if (d_b) hipFree(d_b);
	if (c) free(c);
        if (d_c) hipFree(d_c);

        if (handle) hipblasDestroy(handle);

}

void Save_Result() {

        FILE *pFile;
        int i;

        // Save the matrix
	pFile = fopen("c.txt","w");
        // Save the vector c
        for (i = 0; i < m; i++) {
		fprintf(pFile, "%g\t", c[i]);
		fprintf(pFile, "\n");
        }
        fclose(pFile);
}

