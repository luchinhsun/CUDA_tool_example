#include "hip/hip_runtime.h"
#include <stdio.h>
#include <malloc.h>
#include "head.h"

#define tpb 256

void Allocate(double **subT, double **subNUT, double **subMUT, double **subNDT, double **subMDT, double **d_subT, double **d_subBT, double **d_subNUT, double **d_subMUT, double **d_subNDT, double **d_subMDT){
	hipError_t Error;
	*subT = (double *)malloc(subN*sizeof(double));
	*subNUT = (double *)malloc(n*sizeof(double));
	*subMUT = (double *)malloc(n*sizeof(double));
	*subNDT = (double *)malloc(n*sizeof(double));
	*subMDT = (double *)malloc(n*sizeof(double));
	
	hipMalloc((void**)d_subT,subN*sizeof(double));
	hipMalloc((void**)d_subBT,subBN*sizeof(double));
	hipMalloc((void**)d_subNUT,n*sizeof(double));
	hipMalloc((void**)d_subMUT,n*sizeof(double));
	hipMalloc((void**)d_subNDT,n*sizeof(double));
	Error = hipMalloc((void**)d_subMDT,n*sizeof(double));
	if (DEBUG)	printf("CUDA Error(malloc d_subMDT) = %s\n", hipGetErrorString(Error));
}

void Send_To_Device(int phase, double **subT, double **subNUT, double **subNDT, double **d_subT, double **d_subNUT, double **d_subNDT){
	hipError_t Error;
	if(phase == 0){
		Error = hipMemcpy(*d_subT, *subT, subN*sizeof(double), hipMemcpyHostToDevice);
		if (DEBUG)	printf("CUDA Error(copy subT->d_subT) = %s\n", hipGetErrorString(Error));
	}
	if(phase == 1){
		Error = hipMemcpy(*d_subNUT, *subNUT, n*sizeof(double), hipMemcpyHostToDevice);
		if (DEBUG)	printf("CUDA Error(copy subNUT->d_subNUT) = %s\n", hipGetErrorString(Error));
		Error = hipMemcpy(*d_subNDT, *subNDT, n*sizeof(double), hipMemcpyHostToDevice);
		if (DEBUG)	printf("CUDA Error(copy subNDT->d_subNDT) = %s\n", hipGetErrorString(Error));
	}
}

void Send_To_Host(int phase, double **subT, double **subMUT, double **subMDT, double **d_subT, double **d_subMUT, double **d_subMDT){
	hipError_t Error;
	if(phase == 0){
		Error = hipMemcpy(*subT, *d_subT, subN*sizeof(double), hipMemcpyDeviceToHost);
		if (DEBUG)	printf("CUDA Error(copy d_subT->subT) = %s\n", hipGetErrorString(Error));
	}
	if(phase == 1){
		Error = hipMemcpy(*subMUT, *d_subMUT, n*sizeof(double), hipMemcpyDeviceToHost);
		if (DEBUG)	printf("CUDA Error(copy d_subMUT->subMUT) = %s\n", hipGetErrorString(Error));
		Error = hipMemcpy(*subMDT, *d_subMDT, n*sizeof(double), hipMemcpyDeviceToHost);
		if (DEBUG)	printf("CUDA Error(copy d_subMDT->subMDT) = %s\n", hipGetErrorString(Error));
	}
}

__global__ void boundary0(double *d_subT, double *d_subBT, double*d_subMUT, double *d_subMDT){
	int i = blockDim.x * blockIdx.x +threadIdx.x;
	int x, id;

	if(i<subN){
		x = i/n;
		id = i+(n+2)+1+2*x;
		d_subBT[id] = d_subT[i];
	}
	if(i<subn){
		d_subBT[(i+1)*(n+2)] = d_subT[i*n+n-1];
		d_subBT[(i+1)*(n+2)+n+1] = d_subT[i*n];
	}
	if(i<n){
		d_subMUT[i] = d_subT[i];
		d_subMDT[i] = d_subT[(subn-1)*n+i];
	}
}

__global__ void boundary1(double *d_subBT, double*d_subNUT, double *d_subNDT){
	int i = blockDim.x * blockIdx.x +threadIdx.x;
	
	if(i<n){
		d_subBT[i+1] = d_subNDT[i];
		d_subBT[(subn+1)*(n+2)+i+1] = d_subNUT[i];
	}
}

void CUDA_bdy(int phase, double **d_subT, double **d_subBT, double **d_subNUT, double **d_subMUT, double **d_subNDT, double **d_subMDT){
	int bpg0 = (subN+tpb-1)/tpb;
	int bpg1 = (n+tpb-1)/tpb;
	if(phase == 0)	boundary0<<<bpg0, tpb>>>(*d_subT, *d_subBT, *d_subMUT, *d_subMDT);
	if(phase == 1)	boundary1<<<bpg1, tpb>>>(*d_subBT, *d_subNUT, *d_subNDT);
	hipDeviceSynchronize();
}

__global__ void Forward_Euler(double *d_subT, double *d_subBT){
	int i = blockDim.x * blockIdx.x +threadIdx.x;
	int id, x;
	
	if(i<subN){
		x = i/n;
		id = i+(n+2)+1+2*x;
		d_subT[i] = d_subBT[id] + 0.1*(d_subBT[id-(n+2)] + d_subBT[id+(n+2)] + d_subBT[id-1] + d_subBT[id+1]);
	}
			
}

void CUDA_FE(double **d_subT, double **d_subBT){
	int bpg = (subN+tpb-1)/tpb;
	Forward_Euler<<<bpg, tpb>>>(*d_subT, *d_subBT);
}

void Free(double **subT, double **subNUT, double **subMUT, double **subNDT, double **subMDT, double **d_subT, double **d_subBT, double **d_subNUT, double **d_subMUT, double **d_subNDT, double **d_subMDT){
	
	free(*subT);
	free(*subMUT);free(*subMDT);
	free(*subNUT);free(*subNDT);
	hipFree(*d_subT);hipFree(*d_subBT);
	hipFree(*d_subNUT);hipFree(*d_subMUT);
	hipFree(*d_subNDT);hipFree(*d_subMDT);
}
