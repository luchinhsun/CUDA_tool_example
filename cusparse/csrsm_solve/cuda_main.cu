#include "head.h"

//variable for cusparse
hipsparseStatus_t status;
hipsparseHandle_t handle=0;
hipsparseMatDescr_t descr=0;
hipsparseMatDescr_t descrL=0;
hipsparseMatDescr_t descrU=0;
cusparseSolveAnalysisInfo_t infoA=0;
cusparseSolveAnalysisInfo_t info_u=0;
int *cooRowIndexHostPtr;
int * cooColIndexHostPtr;
float * cooValHostPtr;
int *cooRowIndex;
int * cooColIndex;
float * cooVal;
float * cooValLU;
float * yHostPtr;
float * y;
float * xHostPtr;
float * x;
float * temp;
int * csrRowPtr;
float * A;

float dzero =0.0;
float done =1.0;
float dtwo =2.0;
float dthree=3.0;
float dfive =5.0;

void Allocate_Memory_and_Init(){
	//cusparse
	size_t size = nnz*sizeof(int);
	cooRowIndexHostPtr = (int *) malloc(size);
	cooColIndexHostPtr = (int *) malloc(size);
	cooValHostPtr = (float *)malloc(nnz*sizeof(float));

	cooRowIndexHostPtr[0] = 0;cooColIndexHostPtr[0]=0;cooValHostPtr[0]=-2.0;
	cooRowIndexHostPtr[1] = 0;cooColIndexHostPtr[1]=1;cooValHostPtr[1]=1.0;

	cooRowIndexHostPtr[2] = 1;cooColIndexHostPtr[2]=0;cooValHostPtr[2]=1.0;
	cooRowIndexHostPtr[3] = 1;cooColIndexHostPtr[3]=1;cooValHostPtr[3]=-2.0;
	cooRowIndexHostPtr[4] = 1;cooColIndexHostPtr[4]=2;cooValHostPtr[4]=1.0;
	int i;
	for(i=5;i<(nnz-3);i=i+3){
		cooRowIndexHostPtr[i] = cooRowIndexHostPtr[i-3]+1;	
		cooColIndexHostPtr[i] = cooColIndexHostPtr[i-3]+1;
		cooRowIndexHostPtr[i+1] = cooRowIndexHostPtr[i];	
		cooColIndexHostPtr[i+1] = cooColIndexHostPtr[i]+1;
		cooRowIndexHostPtr[i+2] = cooRowIndexHostPtr[i+1];	
		cooColIndexHostPtr[i+2] = cooColIndexHostPtr[i+1]+1;
		cooValHostPtr[i]=1.0;
		cooValHostPtr[i+1]=-2.0;
		cooValHostPtr[i+2]=1.0;
	}
	cooRowIndexHostPtr[nnz-2] = n-1;cooColIndexHostPtr[nnz-2]=n-2;cooValHostPtr[nnz-2]=1.0;
        cooRowIndexHostPtr[nnz-1] = n-1;cooColIndexHostPtr[nnz-1]=n-1;cooValHostPtr[nnz-1]=-2.0;

	A	= (float *)malloc(n*n*sizeof(float));
	yHostPtr    = (float *)malloc(n*n*sizeof(float));
	for (i=0;i<n*n;i++){
		if(i%n == i/n)	yHostPtr[i] = -1.0;
		else	yHostPtr[i] = 0.0;
	}

	xHostPtr    = (float *)malloc(n*n*sizeof(float));

	hipError_t Error;

	Error = hipMalloc((void**)&cooRowIndex, size);
	printf("CUDA error(malloc RowIndex) = %s\n",hipGetErrorString(Error));
	Error = hipMalloc((void**)&cooColIndex, size);
	printf("CUDA error(malloc ColIndex) = %s\n",hipGetErrorString(Error));
	Error = hipMalloc((void**)&cooVal, nnz*sizeof(float));
	printf("CUDA error(malloc Val) = %s\n",hipGetErrorString(Error));
	Error = hipMalloc((void**)&cooValLU, nnz*sizeof(float));
        printf("CUDA error(malloc Val) = %s\n",hipGetErrorString(Error));
	Error = hipMalloc((void**)&y, n*n*sizeof(float));
	printf("CUDA error(malloc y) = %s\n",hipGetErrorString(Error));
	Error = hipMalloc((void**)&x, n*n*sizeof(float));
        printf("CUDA error(malloc x) = %s\n",hipGetErrorString(Error));
	Error = hipMalloc((void**)&temp, n*n*sizeof(float));
        printf("CUDA error(malloc temp) = %s\n",hipGetErrorString(Error));
	Error = hipMalloc((void**)&csrRowPtr,(n+1)*sizeof(int));
        printf("CUDA error(malloc csrRowPtr) = %s\n",hipGetErrorString(Error));

	status= hipsparseCreate(&handle);
	status= hipsparseCreateMatDescr(&descr);
	hipsparseSetMatType(descr,HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(descr,HIPSPARSE_INDEX_BASE_ZERO);

        status = cusparseCreateSolveAnalysisInfo(&infoA);
	status = cusparseCreateSolveAnalysisInfo(&info_u);

	status = hipsparseCreateMatDescr(&descrL);
	hipsparseSetMatType(descrL,HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(descrL,HIPSPARSE_INDEX_BASE_ZERO);
	hipsparseSetMatFillMode(descrL, HIPSPARSE_FILL_MODE_LOWER);
    	hipsparseSetMatDiagType(descrL, HIPSPARSE_DIAG_TYPE_UNIT);

    	status = hipsparseCreateMatDescr(&descrU);
    	hipsparseSetMatType(descrU,HIPSPARSE_MATRIX_TYPE_GENERAL);
    	hipsparseSetMatIndexBase(descrU,HIPSPARSE_INDEX_BASE_ZERO);
    	hipsparseSetMatFillMode(descrU, HIPSPARSE_FILL_MODE_UPPER);
    	hipsparseSetMatDiagType(descrU, HIPSPARSE_DIAG_TYPE_NON_UNIT);
}

void Send_To_Device(){
	hipError_t Error;
	size_t size = nnz*sizeof(int);
	Error = hipMemcpy(cooRowIndex, cooRowIndexHostPtr, size, hipMemcpyHostToDevice);
	printf("CUDA error(memcpy RowIndex) = %s\n",hipGetErrorString(Error));
	Error = hipMemcpy(cooColIndex, cooColIndexHostPtr, size, hipMemcpyHostToDevice);
	printf("CUDA error(memcpy ColIndex) = %s\n",hipGetErrorString(Error));
	Error = hipMemcpy(cooVal, cooValHostPtr, (size_t)(nnz*sizeof(float)), hipMemcpyHostToDevice);
	printf("CUDA error(memcpy Val) = %s\n",hipGetErrorString(Error));
	Error = hipMemcpy(y, yHostPtr, (size_t)(n*n*sizeof(float)), hipMemcpyHostToDevice);
	printf("CUDA error(memcpy y) = %s\n",hipGetErrorString(Error));
	Error = hipMemcpy(x, xHostPtr, (size_t)(n*n*sizeof(float)), hipMemcpyHostToDevice);
        printf("CUDA error(memcpy x) = %s\n",hipGetErrorString(Error));
}

void Call_GPUFunction(){
	status= hipsparseXcoo2csr(handle,cooRowIndex,nnz,n, csrRowPtr,HIPSPARSE_INDEX_BASE_ZERO);
	if (status != HIPSPARSE_STATUS_SUCCESS) {
		printf("coo2csr fail");
	}

	status= cusparseScsrsm_analysis(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, n, nnz, descr,
                                                cooVal, csrRowPtr, cooColIndex, infoA);
        hipMemcpy(cooValLU, cooVal, nnz*sizeof(float), hipMemcpyDeviceToDevice);
        status = cusparseScsrilu0(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, n, descr,
                                                cooValLU, csrRowPtr, cooColIndex, infoA);
        status = cusparseScsrsm_analysis(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, n, nnz, descrU,
                                                cooVal, csrRowPtr, cooColIndex, info_u);
        status = cusparseScsrsm_solve(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, n, n, &done, descrL,
                                              cooValLU, csrRowPtr, cooColIndex, infoA, y, n, temp, n);
        status = cusparseScsrsm_solve(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, n, n, &done, descrU,
                                              cooValLU, csrRowPtr, cooColIndex, info_u, temp, n, x, n);
}


void Send_To_Host(){
	hipError_t Error;
	Error = hipMemcpy(yHostPtr, y, (size_t)(n*n*sizeof(float)), hipMemcpyDeviceToHost);
	printf("CUDA error(memcpy y->yHostPtr) = %s\n",hipGetErrorString(Error));
	Error = hipMemcpy(xHostPtr, x, (size_t)(n*n*sizeof(float)), hipMemcpyDeviceToHost);
        printf("CUDA error(memcpy x->xHostPtr) = %s\n",hipGetErrorString(Error));
}

void Free_Memory(){
	if (yHostPtr) free(yHostPtr);
	if (xHostPtr) free(xHostPtr);
        if (cooRowIndexHostPtr) free(cooRowIndexHostPtr);
        if (cooColIndexHostPtr) free(cooColIndexHostPtr);
        if (cooValHostPtr) free(cooValHostPtr);
        if (y) hipFree(y);
	if (x) hipFree(x);
	if (temp) hipFree(temp);
        if (csrRowPtr) hipFree(csrRowPtr);
        if (cooRowIndex) hipFree(cooRowIndex);
        if (cooColIndex) hipFree(cooColIndex);
        if (cooVal) hipFree(cooVal);
	if (cooValLU) hipFree(cooValLU);
        if (descr) hipsparseDestroyMatDescr(descr);
        if (handle) hipsparseDestroy(handle);
	if (descrL) hipsparseDestroyMatDescr(descrL);
        if (descrU) hipsparseDestroyMatDescr(descrU);
        if (A) free(A);

	cusparseDestroySolveAnalysisInfo(infoA);
        cusparseDestroySolveAnalysisInfo(info_u);

}

void Save_Result() {

        FILE *pFile;
        int i, j;

        // Save the matrix A
	for(i=0;i<n*n;i++){
                A[i] = 0.0;
        }
        for(i=0;i<nnz;i++){
                A[cooRowIndexHostPtr[i]*n+cooColIndexHostPtr[i]] = cooValHostPtr[i];
        }
        pFile = fopen("A.txt","w");
        for (i = 0; i < n; i++) {
                for (j = 0; j < n; j++) {
                        fprintf(pFile, "%g\t", A[i*n+j]);
                }
                fprintf(pFile, "\n");
        }
        fclose(pFile);

	pFile = fopen("B.txt","w");
        // Save the matrix B
        for (i = 0; i < n; i++) {
		for (j = 0; j<n; ++j){
                	fprintf(pFile, "%g\t", yHostPtr[i*n+j]);
		}
		fprintf(pFile, "\n");
        }
        fclose(pFile);

        pFile = fopen("X.txt","w");
        // Save the matrix x
        for (i = 0; i < n; i++) {
		for (j = 0; j<n; ++j){
                	fprintf(pFile, "%g\t", xHostPtr[i*n+j]);
		}
                fprintf(pFile, "\n");
        }
        fclose(pFile);
}
