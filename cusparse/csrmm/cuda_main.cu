#include "head.h"

//variable for cusparse
hipsparseStatus_t status;
hipsparseHandle_t handle=0;
hipsparseMatDescr_t descr=0;
int *cooRowIndexHostPtr;
int * cooColIndexHostPtr;
float * cooValHostPtr;
int *cooRowIndex;
int * cooColIndex;
float * cooVal;
float * cooValLU;
float * yHostPtr;
float * y;
float * xHostPtr;
float * x;
float * temp;
int * csrRowPtr;
float * A;

float dzero =0.0;
float done =1.0;
float dtwo =2.0;
float dthree=3.0;
float dfive =5.0;

void Allocate_Memory_and_Init(){
	//cusparse
	size_t size = nnz*sizeof(int);
	cooRowIndexHostPtr = (int *) malloc(size);
	cooColIndexHostPtr = (int *) malloc(size);
	cooValHostPtr = (float *)malloc(nnz*sizeof(float));

	cooRowIndexHostPtr[0] = 0;cooColIndexHostPtr[0]=1;cooValHostPtr[0]=1.0;
	cooRowIndexHostPtr[1] = 0;cooColIndexHostPtr[1]=2;cooValHostPtr[1]=2.0;
	cooRowIndexHostPtr[2] = 1;cooColIndexHostPtr[2]=0;cooValHostPtr[2]=3.0;
	cooRowIndexHostPtr[3] = 1;cooColIndexHostPtr[3]=1;cooValHostPtr[3]=4.0;
	cooRowIndexHostPtr[4] = 1;cooColIndexHostPtr[4]=2;cooValHostPtr[4]=5.0;
	cooRowIndexHostPtr[5] = 2;cooColIndexHostPtr[5]=1;cooValHostPtr[5]=6.0;
	cooRowIndexHostPtr[6] = 2;cooColIndexHostPtr[6]=3;cooValHostPtr[6]=7.0;
	cooRowIndexHostPtr[7] = 3;cooColIndexHostPtr[7]=1;cooValHostPtr[7]=8.0;
	cooRowIndexHostPtr[8] = 4;cooColIndexHostPtr[8]=2;cooValHostPtr[8]=9.0;

	int i;
	A	= (float *)malloc(m*k*sizeof(float));
	yHostPtr    = (float *)malloc(k*n*sizeof(float));
	for (i=0;i<k*n;i++){
		yHostPtr[i] = i;
	}

	xHostPtr    = (float *)malloc(m*n*sizeof(float));

	hipError_t Error;

	Error = hipMalloc((void**)&cooRowIndex, size);
	printf("CUDA error(malloc RowIndex) = %s\n",hipGetErrorString(Error));
	Error = hipMalloc((void**)&cooColIndex, size);
	printf("CUDA error(malloc ColIndex) = %s\n",hipGetErrorString(Error));
	Error = hipMalloc((void**)&cooVal, nnz*sizeof(float));
	printf("CUDA error(malloc Val) = %s\n",hipGetErrorString(Error));
	Error = hipMalloc((void**)&cooValLU, nnz*sizeof(float));
        printf("CUDA error(malloc Val) = %s\n",hipGetErrorString(Error));
	Error = hipMalloc((void**)&y, k*n*sizeof(float));
	printf("CUDA error(malloc y) = %s\n",hipGetErrorString(Error));
	Error = hipMalloc((void**)&x, m*n*sizeof(float));
        printf("CUDA error(malloc x) = %s\n",hipGetErrorString(Error));
	Error = hipMalloc((void**)&csrRowPtr,(m+1)*sizeof(int));
        printf("CUDA error(malloc csrRowPtr) = %s\n",hipGetErrorString(Error));

	status= hipsparseCreate(&handle);
	status= hipsparseCreateMatDescr(&descr);
	hipsparseSetMatType(descr,HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(descr,HIPSPARSE_INDEX_BASE_ZERO);
}

void Send_To_Device(){
	hipError_t Error;
	size_t size = nnz*sizeof(int);
	Error = hipMemcpy(cooRowIndex, cooRowIndexHostPtr, size, hipMemcpyHostToDevice);
	printf("CUDA error(memcpy RowIndex) = %s\n",hipGetErrorString(Error));
	Error = hipMemcpy(cooColIndex, cooColIndexHostPtr, size, hipMemcpyHostToDevice);
	printf("CUDA error(memcpy ColIndex) = %s\n",hipGetErrorString(Error));
	Error = hipMemcpy(cooVal, cooValHostPtr, (size_t)(nnz*sizeof(float)), hipMemcpyHostToDevice);
	printf("CUDA error(memcpy Val) = %s\n",hipGetErrorString(Error));
	Error = hipMemcpy(y, yHostPtr, (size_t)(k*n*sizeof(float)), hipMemcpyHostToDevice);
	printf("CUDA error(memcpy y) = %s\n",hipGetErrorString(Error));
	Error = hipMemcpy(x, xHostPtr, (size_t)(m*n*sizeof(float)), hipMemcpyHostToDevice);
        printf("CUDA error(memcpy x) = %s\n",hipGetErrorString(Error));
}

void Call_GPUFunction(){
	status= hipsparseXcoo2csr(handle,cooRowIndex,nnz,m, csrRowPtr,HIPSPARSE_INDEX_BASE_ZERO);
	if (status != HIPSPARSE_STATUS_SUCCESS) {
		printf("coo2csr fail");
	}

	status= hipsparseScsrmm(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, m, n, k, nnz, &done, descr,
                                                cooVal, csrRowPtr, cooColIndex, y, k, &done, x, m);
}


void Send_To_Host(){
	hipError_t Error;
	Error = hipMemcpy(yHostPtr, y, (size_t)(k*n*sizeof(float)), hipMemcpyDeviceToHost);
	printf("CUDA error(memcpy y->yHostPtr) = %s\n",hipGetErrorString(Error));
	Error = hipMemcpy(xHostPtr, x, (size_t)(m*n*sizeof(float)), hipMemcpyDeviceToHost);
        printf("CUDA error(memcpy x->xHostPtr) = %s\n",hipGetErrorString(Error));
}

void Free_Memory(){
	if (yHostPtr) free(yHostPtr);
	if (xHostPtr) free(xHostPtr);
        if (cooRowIndexHostPtr) free(cooRowIndexHostPtr);
        if (cooColIndexHostPtr) free(cooColIndexHostPtr);
        if (cooValHostPtr) free(cooValHostPtr);
        if (y) hipFree(y);
	if (x) hipFree(x);
	if (temp) hipFree(temp);
        if (csrRowPtr) hipFree(csrRowPtr);
        if (cooRowIndex) hipFree(cooRowIndex);
        if (cooColIndex) hipFree(cooColIndex);
        if (cooVal) hipFree(cooVal);
	if (cooValLU) hipFree(cooValLU);
        if (descr) hipsparseDestroyMatDescr(descr);
        if (handle) hipsparseDestroy(handle);
        if (A) free(A);
}

void Save_Result() {

        FILE *pFile;
        int i, j;

        // Save the matrix A
	for(i=0;i<m*k;i++){
                A[i] = 0.0;
        }
        for(i=0;i<nnz;i++){
                A[cooRowIndexHostPtr[i]*k+cooColIndexHostPtr[i]] = cooValHostPtr[i];
        }
        pFile = fopen("A.txt","w");
        for (i = 0; i < m; i++) {
                for (j = 0; j < k; j++) {
                        fprintf(pFile, "%g\t", A[i*k+j]);
                }
                fprintf(pFile, "\n");
        }
        fclose(pFile);

	pFile = fopen("B.txt","w");
        // Save the matrix B
        for (i = 0; i < k; i++){
		for (j = 0; j<n; j++){
                	fprintf(pFile, "%g\t", yHostPtr[i+j*k]);
		}
		fprintf(pFile, "\n");
        }
        fclose(pFile);

        pFile = fopen("X.txt","w");
        // Save the matrix x
        for (i = 0; i < m; i++) {
		for (j = 0; j<n; ++j){
                	fprintf(pFile, "%g\t", xHostPtr[i+j*m]);
		}
                fprintf(pFile, "\n");
        }
        fclose(pFile);
}
