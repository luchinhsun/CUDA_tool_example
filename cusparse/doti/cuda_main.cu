#include "head.h"

//variable for cusparse
hipsparseStatus_t status;
hipsparseHandle_t handle=0;

float * yHostPtr;
float * y;
float * xHostPtr;
float * x;
int * xHostInd;
int * xInd;
float * result;
float * Hostresult;

void Allocate_Memory_and_Init(){
	//cusparse
	size_t size = nnz*sizeof(int);

	Hostresult    = (float *)malloc(1*sizeof(float));
	yHostPtr    = (float *)malloc(n*sizeof(float));
	yHostPtr[0] = 50.0; yHostPtr[1] = 60.0; yHostPtr[2] = 70.0; yHostPtr[3] = 80.0;
	xHostPtr    = (float *)malloc(nnz*sizeof(float));
	xHostPtr[0] = 20.0; xHostPtr[1] = 11.0;// xHostPtr[2] = 0.0; xHostPtr[3] = 0.0;
	xHostInd = (int *) malloc(size);
	xHostInd[0] = 1.0; xHostInd[1] = 3.0;

	hipError_t Error;

	Error = hipMalloc((void**)&y, n*sizeof(float));
	printf("CUDA error(malloc y) = %s\n",hipGetErrorString(Error));
	Error = hipMalloc((void**)&x, nnz*sizeof(float));
        printf("CUDA error(malloc x) = %s\n",hipGetErrorString(Error));
	Error = hipMalloc((void**)&xInd, nnz*sizeof(int));
        printf("CUDA error(malloc xInd) = %s\n",hipGetErrorString(Error));
	Error = hipMalloc((void**)&result, 1*sizeof(float));
        printf("CUDA error(malloc result) = %s\n",hipGetErrorString(Error));

	status= hipsparseCreate(&handle);
}

void Send_To_Device(){
	hipError_t Error;
	size_t size = nnz*sizeof(int);

	Error = hipMemcpy(xInd, xHostInd, size, hipMemcpyHostToDevice);
        printf("CUDA error(memcpy xInd) = %s\n",hipGetErrorString(Error));
	Error = hipMemcpy(y, yHostPtr, (size_t)(n*sizeof(float)), hipMemcpyHostToDevice);
	printf("CUDA error(memcpy y) = %s\n",hipGetErrorString(Error));
	Error = hipMemcpy(x, xHostPtr, (size_t)(n*sizeof(float)), hipMemcpyHostToDevice);
        printf("CUDA error(memcpy x) = %s\n",hipGetErrorString(Error));
}

void Call_GPUFunction(){
	hipsparseSetPointerMode(handle, HIPSPARSE_POINTER_MODE_DEVICE);
	hipsparseSdoti(handle, nnz, x, xInd, y, result, HIPSPARSE_INDEX_BASE_ZERO);

}


void Send_To_Host(){
	hipError_t Error;

	Error = hipMemcpy(Hostresult, result, (size_t)(1*sizeof(float)), hipMemcpyDeviceToHost);
        printf("CUDA error(memcpy x->xHostPtr) = %s\n",hipGetErrorString(Error));
}

void Free_Memory(){
	if (yHostPtr) free(yHostPtr);
	if (xHostPtr) free(xHostPtr);
        if (y) hipFree(y);
	if (x) hipFree(x);

        if (handle) hipsparseDestroy(handle);

	if (Hostresult) free(Hostresult);
	if (result) hipFree(result);
	if (xHostInd) free(xHostInd);
	if (xInd) hipFree(xInd);
}
