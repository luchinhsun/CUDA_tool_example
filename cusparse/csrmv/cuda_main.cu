#include "head.h"

//variable for cusparse
hipsparseStatus_t status;
hipsparseHandle_t handle=0;
hipsparseMatDescr_t descr=0;
hipsparseMatDescr_t descrL=0;
hipsparseMatDescr_t descrU=0;
int *cooRowIndexHostPtr;
int * cooColIndexHostPtr;
float * cooValHostPtr;
int *cooRowIndex;
int * cooColIndex;
float * cooVal;
float * cooValLU;
float * yHostPtr;
float * y;
float * xHostPtr;
float * x;
float * temp;
int * csrRowPtr;

float dzero =0.0;
float done =1.0;
float dtwo =2.0;
float dthree=3.0;
float dfive =5.0;

void Allocate_Memory_and_Init(){
	//cusparse_Init
	int i;
	size_t size = nnz*sizeof(int);
	cooRowIndexHostPtr = (int *) malloc(size);
	cooColIndexHostPtr = (int *) malloc(size);
	cooValHostPtr = (float *)malloc(nnz*sizeof(float));
	cooRowIndexHostPtr[0]=0; cooColIndexHostPtr[0]=0; cooValHostPtr[0]=1.0;
	cooRowIndexHostPtr[1]=0; cooColIndexHostPtr[1]=2; cooValHostPtr[1]=2.0;
	cooRowIndexHostPtr[2]=0; cooColIndexHostPtr[2]=3; cooValHostPtr[2]=3.0;
	cooRowIndexHostPtr[3]=1; cooColIndexHostPtr[3]=1; cooValHostPtr[3]=4.0;
	cooRowIndexHostPtr[4]=2; cooColIndexHostPtr[4]=0; cooValHostPtr[4]=5.0;
	cooRowIndexHostPtr[5]=2; cooColIndexHostPtr[5]=2; cooValHostPtr[5]=6.0;
	cooRowIndexHostPtr[6]=2; cooColIndexHostPtr[6]=3; cooValHostPtr[6]=7.0;
	cooRowIndexHostPtr[7]=3; cooColIndexHostPtr[7]=1; cooValHostPtr[7]=8.0;
	cooRowIndexHostPtr[8]=3; cooColIndexHostPtr[8]=3; cooValHostPtr[8]=9.0;
	cooRowIndexHostPtr[9]=4; cooColIndexHostPtr[9]=0; cooValHostPtr[9]=10.0;
        cooRowIndexHostPtr[10]=4; cooColIndexHostPtr[10]=3; cooValHostPtr[10]=11.0;

	yHostPtr    = (float *)malloc(n*sizeof(float));
	yHostPtr[0] = 50.0; yHostPtr[1] = 60.0; yHostPtr[2] = 70.0; yHostPtr[3] = 80.0;
	xHostPtr    = (float *)malloc(m*sizeof(float));
	for(i=0;i<m;i++){
		xHostPtr[i] = 0.0;
	}

	hipError_t Error;

	Error = hipMalloc((void**)&cooRowIndex, size);
	printf("CUDA error(malloc RowIndex) = %s\n",hipGetErrorString(Error));
	Error = hipMalloc((void**)&cooColIndex, size);
	printf("CUDA error(malloc ColIndex) = %s\n",hipGetErrorString(Error));
	Error = hipMalloc((void**)&cooVal, nnz*sizeof(float));
	printf("CUDA error(malloc Val) = %s\n",hipGetErrorString(Error));
	Error = hipMalloc((void**)&cooValLU, nnz*sizeof(float));
        printf("CUDA error(malloc Val) = %s\n",hipGetErrorString(Error));
	Error = hipMalloc((void**)&y, n*sizeof(float));
	printf("CUDA error(malloc y) = %s\n",hipGetErrorString(Error));
	Error = hipMalloc((void**)&x, m*sizeof(float));
        printf("CUDA error(malloc x) = %s\n",hipGetErrorString(Error));
	Error = hipMalloc((void**)&csrRowPtr,(m+1)*sizeof(int));
        printf("CUDA error(malloc csrRowPtr) = %s\n",hipGetErrorString(Error));

	status= hipsparseCreate(&handle);
	status= hipsparseCreateMatDescr(&descr);

	hipsparseSetMatType(descr,HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(descr,HIPSPARSE_INDEX_BASE_ZERO);
}

void Send_To_Device(){
	hipError_t Error;
	size_t size = nnz*sizeof(int);
	Error = hipMemcpy(cooRowIndex, cooRowIndexHostPtr, size, hipMemcpyHostToDevice);
	printf("CUDA error(memcpy RowIndex) = %s\n",hipGetErrorString(Error));
	Error = hipMemcpy(cooColIndex, cooColIndexHostPtr, size, hipMemcpyHostToDevice);
	printf("CUDA error(memcpy ColIndex) = %s\n",hipGetErrorString(Error));
	Error = hipMemcpy(cooVal, cooValHostPtr, (size_t)(nnz*sizeof(float)), hipMemcpyHostToDevice);
	printf("CUDA error(memcpy Val) = %s\n",hipGetErrorString(Error));
	Error = hipMemcpy(y, yHostPtr, (size_t)(n*sizeof(float)), hipMemcpyHostToDevice);
	printf("CUDA error(memcpy y) = %s\n",hipGetErrorString(Error));
	Error = hipMemcpy(x, xHostPtr, (size_t)(m*sizeof(float)), hipMemcpyHostToDevice);
        printf("CUDA error(memcpy x) = %s\n",hipGetErrorString(Error));
}

void Call_GPUFunction(){
	status= hipsparseXcoo2csr(handle,cooRowIndex, nnz, m, csrRowPtr,HIPSPARSE_INDEX_BASE_ZERO);
	if (status != HIPSPARSE_STATUS_SUCCESS) {
		printf("coo2csr fail");
	}

	status= hipsparseScsrmv(handle,HIPSPARSE_OPERATION_NON_TRANSPOSE, m, n, nnz,
				&done, descr, cooVal, csrRowPtr, cooColIndex, &y[0], &dzero, &x[0]);
	if (status != HIPSPARSE_STATUS_SUCCESS) {
		printf("csrmv fail");
	}
}


void Send_To_Host(){
	hipError_t Error;
	Error = hipMemcpy(yHostPtr, y, (size_t)(n*sizeof(float)), hipMemcpyDeviceToHost);
	printf("CUDA error(memcpy y->yHostPtr) = %s\n",hipGetErrorString(Error));
	Error = hipMemcpy(xHostPtr, x, (size_t)(m*sizeof(float)), hipMemcpyDeviceToHost);
        printf("CUDA error(memcpy x->xHostPtr) = %s\n",hipGetErrorString(Error));
}

void Free_Memory(){
	if (yHostPtr) free(yHostPtr);
	if (xHostPtr) free(xHostPtr);
        if (cooRowIndexHostPtr) free(cooRowIndexHostPtr);
        if (cooColIndexHostPtr) free(cooColIndexHostPtr);
        if (cooValHostPtr) free(cooValHostPtr);
        if (y) hipFree(y);
	if (x) hipFree(x);
        if (csrRowPtr) hipFree(csrRowPtr);
        if (cooRowIndex) hipFree(cooRowIndex);
        if (cooColIndex) hipFree(cooColIndex);
        if (cooVal) hipFree(cooVal);
	if (cooValLU) hipFree(cooValLU);
        if (descr) hipsparseDestroyMatDescr(descr);
        if (handle) hipsparseDestroy(handle);

}
